#include "hip/hip_runtime.h"
#include "cuda-kernel.hpp"
#include <stdio.h>

#define THREAD_WORKLOAD 100 
#define THREAD_PREBLOCK 32
__global__ void runKernel (Bit *transactions, int transaction_size, 
                           Bit *itemSets, int itemSetsSize, int unit_len,
                           int *cuda_result) {
    int size = (transaction_size + THREAD_PREBLOCK - 1) / THREAD_PREBLOCK;
    int start = threadIdx.x * size;
    if (threadIdx.x == THREAD_PREBLOCK - 1)
        size = transaction_size - (THREAD_PREBLOCK - 1) * size;
    int item_start = blockIdx.x * unit_len;

    int result = 0;
    for (int i = 0; i < size; i++) {
        Bit *tran = &transactions[(start + i) * unit_len];
        Bit flag = 0;
        for (int j = 0; j < unit_len; j++) {
            flag |= (itemSets[item_start + j] & tran[j]) ^ itemSets[item_start + j]; 
        }
        if (!flag)
            result++;
    }

    int r = atomicAdd(&cuda_result[blockIdx.x], result);
}

void getSupport (Bit *transactions, int transaction_size, Bit *itemSets,
                 int itemSetsSize, int unit_len, int *result) {
    dim3 dimGrid(itemSetsSize);
    dim3 dimBlock(THREAD_PREBLOCK);
    int *cuda_result;
    
    for (int i = 0; i < itemSetsSize; i++)
        result[i] = 0;
    hipMalloc((void **)&cuda_result, sizeof(int) * itemSetsSize);
    hipMemcpy(cuda_result, result, itemSetsSize * sizeof(int), hipMemcpyHostToDevice);

    runKernel<<<dimGrid, dimBlock>>>(transactions, transaction_size, itemSets,
                                     itemSetsSize, unit_len, cuda_result);

    hipMemcpy(result, cuda_result, itemSetsSize * sizeof(int), hipMemcpyDeviceToHost);
}
